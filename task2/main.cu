/**
 * Nasobeni matic v globalni pameti.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <iostream>

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {                                                                          \
                                                                                                            \
	hipError_t err = value;                                                                                \
                                                                                                            \
	if (err != hipSuccess) {                                                                               \
		fprintf(stderr, "Error %s at line %d in file %s\n", hipGetErrorString(err), __LINE__, __FILE__);   \
		exit(1);                                                                                            \
	}                                                                                                       \
}                                                                                                           \

#define BLOCK_SIZE (16u)

/**
 * Host function that prepares data array and passes it to the CUDA kernel.
 */
int main(int argc, char **argv) {
	
	return 0;
}
